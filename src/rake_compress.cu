#include "hip/hip_runtime.h"
#ifndef RAKE_COMPRESS
#define RAKE_COMPRESS

#include "CSR_matrix.cu"
#include "rake.cu"
#include "compress.cu"



Vertex *GenerateCompressedGraph(const CSR_mat g){

    const Vertex vertices = g.Get_Vertex_count();
    const Vertex edges = g.Get_edge_count();
    Vertex round = 1;
    Vertex *d_host = (Vertex*)malloc(sizeof(Vertex)*vertices);
    

    int *l=(int *)malloc(sizeof(int)*vertices);
    memset(l,0,sizeof(int)*vertices);

    int *c=(int *)malloc(sizeof(int)*vertices);
    memset(c,0,sizeof(int)*vertices);

    Vertex *d_gpu;
    hipMalloc(&d_gpu,sizeof(Vertex)*vertices);
    hipMemset(d_gpu, 0, sizeof(Vertex)*vertices);

    int *l_gpu;
    hipMalloc(&l_gpu,sizeof(int)*vertices);
    hipMemset(l_gpu,0,sizeof(int)*vertices);

    int *c_gpu;
    hipMalloc(&c_gpu,sizeof(int)*vertices);
    hipMemset(c_gpu,0,sizeof(int)*vertices);

    bool *update_gpu;
    hipMalloc(&update_gpu,sizeof(bool));
    hipMemset(update_gpu, 0, sizeof(bool));

    bool *update_host = (bool*)malloc(sizeof(bool));
    *update_host=true;

    CSR_mat intermediate_graph;
    intermediate_graph.Allocate(g.vertex_count, g.edge_count,false);

    CSR_mat host_graph;
    host_graph.Allocate(g.vertex_count,g.edge_count);


    

    while(*update_host){
        hipMemset(update_gpu, false, sizeof(bool));

        //rake process
        find_leaf_kernel<<<(((vertices+1023)/1024)),1024>>>(l_gpu,vertices,d_gpu,g.cct,g.idx);
        hipMemcpy(l, l_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);
        Compute<<<((vertices+1023)/1024),1024>>>(round,g.cct,g.idx,l_gpu,d_gpu,vertices,update_gpu);
        // rake complete

        // printf("rake complete");

        //compress operation

        // marking 2 degree nodes
        find_2_degree_kernel<<<(((vertices+1023)/1024)),1024>>>(c_gpu,vertices,d_gpu,g.cct,g.idx);
        hipMemcpy(c, c_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);
        hipMemcpy(d_host, d_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);
        // marking 2 degree nodes completed

        // copying the graph into cpu
        copy_csr_mat(g,host_graph,true);
        // copying complete

        // marking compressible vertices
        mark_compressed_vertices(host_graph,c,d_host,round);
        hipMemcpy(c_gpu, c, sizeof(Vertex)*vertices, hipMemcpyHostToDevice);
        hipMemcpy(d_gpu, d_host, sizeof(Vertex)*vertices, hipMemcpyHostToDevice);
        // marking done

        // printf("two degree found");

        // storing the intermediate state
        copy_csr_mat(g,intermediate_graph);
        // storing complete

        // printf("vertices marked");
        compress<<<(((vertices+1023)/1024)),1024>>>(intermediate_graph.cct,intermediate_graph.idx,intermediate_graph.nnz,
        g.cct,g.idx,g.nnz,
        d_gpu,vertices,c_gpu,update_gpu);

        // compress operation complete
        round++;
        hipMemcpy(update_host, update_gpu, sizeof(bool), hipMemcpyDeviceToHost);
    }


    hipMemcpy(d_host, d_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);

    hipFree(d_gpu);
    hipFree(l_gpu);
    hipFree(update_gpu);
   
    return d_host;

    

}
#endif