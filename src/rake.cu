#include "hip/hip_runtime.h"
#ifndef RAKE_COPY
#define RAKE_COPY

#include "CSR_matrix.cu"





__global__ void find_leaf_kernel(int *l,const Vertex vertices,const Vertex*d, const Vertex*cct, const Vertex*idx){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid<vertices && d[tid]==0){

        int n_neighbours=0;

        for (int i = cct[tid]; i < cct[tid+1]; i++)
        {
            if(d[idx[i]]==0){
                n_neighbours+=1;
            }
            if(n_neighbours>2){
                break;
            }
        }

        l[tid] = (n_neighbours==1);
        
        
        
    }
}

__global__ void find_binary_nodes(bool *l, const Vertex vertices, const Vertex*d, const Vertex*cct, const Vertex*idx){
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if(tid<vertices && d[tid]==0){
        int n_neighbours=0;
        for (int i = cct[tid]; i < cct[tid+1]; i++)
        {
            n_neighbours+=(d[idx[i]]==0);
            if(n_neighbours>2){
                break;
            }
        }

        l[tid] = (n_neighbours==2);
        
    }
}



void mark_eligible_binary_nodes(bool *l,bool* eligible,const CSR_mat &g){
    Vertex vertices = g.Get_Vertex_count();
    bool*visited = (bool*)malloc(sizeof(bool)*vertices);
    memset(visited, 0,sizeof(bool)*vertices);

    for (int i = 0; i < vertices; i++)
    {
        if(l[i] &&  (!visited[i])&& (!visited[g.idx[g.cct[i]]]) && (!visited[g.idx[g.cct[i]+1]])){
            eligible[i]=1;
            visited[i]=1;
            visited[g.idx[g.cct[i]]]=1;
            visited[g.idx[g.cct[i]+1]]=1;
        }
    }
    
}


__global__ void Compute(const Vertex round,const Vertex*cct,const Vertex*idx,const int*l,Vertex*d, const Vertex vertices, bool*update){
    int tid = blockDim.x*blockIdx.x + threadIdx.x;

    if(tid<vertices && d[tid]==0){

        int n_neighbours=0;

        for (int i = cct[tid]; i < cct[tid+1]; i++)
        {
            if(d[idx[i]]==0){

                n_neighbours++;
                if(l[idx[i]]==1){
                    if((l[tid]==1)){

                        if(tid<idx[i]){

                            d[tid]=round+1;
                            d[idx[i]]=round;
                        }

                        else{
                            d[tid]=round;
                            d[idx[i]]=round+1;
                        }
                    }
                    else{
                        d[idx[i]]=round;
                    }
                    *update=true;
                    break;
                }

            }
        }

        
    }

}

int* getDegree (const CSR_mat g){
    const Vertex vertices = g.Get_Vertex_count();

     int *l=(int *)malloc(sizeof(int)*vertices);
    memset(l,0,sizeof(int)*vertices);

    Vertex *d_gpu;
    hipMalloc(&d_gpu,sizeof(Vertex)*vertices);
    hipMemset(d_gpu, 0, sizeof(Vertex)*vertices);

    int *l_gpu;
    hipMalloc(&l_gpu,sizeof(int)*vertices);
    hipMemset(l_gpu,0,sizeof(int)*vertices);

    find_leaf_kernel<<<(((vertices+1023)/1024)),1024>>>(l_gpu,vertices,d_gpu,g.cct,g.idx);

    hipMemcpy(l, l_gpu, sizeof(int)*vertices, hipMemcpyDeviceToHost);
    return l;

}



Vertex *GenerateCompressedGraph(const CSR_mat g){

    const Vertex vertices = g.Get_Vertex_count();
    const Vertex edges = g.Get_edge_count();
    Vertex round = 1;
    Vertex *d_host = (Vertex*)malloc(sizeof(Vertex)*vertices);
    

    int *l=(int *)malloc(sizeof(int)*vertices);
    memset(l,0,sizeof(int)*vertices);

    Vertex *d_gpu;
    hipMalloc(&d_gpu,sizeof(Vertex)*vertices);
    hipMemset(d_gpu, 0, sizeof(Vertex)*vertices);

    int *l_gpu;
    hipMalloc(&l_gpu,sizeof(int)*vertices);
    hipMemset(l_gpu,0,sizeof(int)*vertices);

    bool *update_gpu;
    hipMalloc(&update_gpu,sizeof(bool));
    hipMemset(update_gpu, 0, sizeof(bool));

    bool *update_host = (bool*)malloc(sizeof(bool));
    *update_host=true;

    

    while(*update_host){
        hipMemset(update_gpu, false, sizeof(bool));
        find_leaf_kernel<<<(((vertices+1023)/1024)),1024>>>(l_gpu,vertices,d_gpu,g.cct,g.idx);
        hipMemcpy(l, l_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);
        Compute<<<((vertices+1023)/1024),1024>>>(round,g.cct,g.idx,l_gpu,d_gpu,vertices,update_gpu);
        round++;
        hipMemcpy(update_host, update_gpu, sizeof(bool), hipMemcpyDeviceToHost);
    }


    hipMemcpy(d_host, d_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);

    hipFree(d_gpu);
    hipFree(l_gpu);
    hipFree(update_gpu);
   
    return d_host;

    

}
#endif
