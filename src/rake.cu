#include "hip/hip_runtime.h"
#ifndef RAKE_COPY
#define RAKE_COPY

#include "CSR_matrix.cu"





__global__ void find_leaf_kernel(bool *l,const Vertex vertices,const Vertex*d, const Vertex*cct, const Vertex*idx){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid<vertices && d[tid]==0){

        int n_neighbours=0;

        for (int i = cct[tid]; i < cct[tid+1]; i++)
        {
            if(d[idx[i]]==0){
                n_neighbours+=1;
            }
            if(n_neighbours>1){
                break;
            }
        }

        if(n_neighbours==1){
            l[tid]=true;
        }
        else{
            l[tid]=false;
        }
        
        
        
    }
}


__global__ void Compute(const Vertex round,const Vertex*cct,const Vertex*idx,const bool*l,Vertex*d, const Vertex vertices, bool*update){
    int tid = blockDim.x*blockIdx.x + threadIdx.x;

    if(tid<vertices && d[tid]==0){

        int n_neighbours=0;

        for (int i = cct[tid]; i < cct[tid+1]; i++)
        {
            if(d[idx[i]]==0){

                n_neighbours++;
                if(l[idx[i]]){
                    if((l[tid])){

                        if(tid<idx[i]){

                            d[tid]=round+1;
                            d[idx[i]]=round;
                        }

                        else{
                            d[tid]=round;
                            d[idx[i]]=round+1;
                        }
                    }
                    else{
                        d[idx[i]]=round;
                    }
                    *update=true;
                    break;
                }

            }
        }

        
    }

}




Vertex *GenerateCompressedGraph(const CSR_mat g){

    const Vertex vertices = g.Get_Vertex_count();
    const Vertex edges = g.Get_edge_count();
    Vertex round = 1;
    Vertex *d_host = (Vertex*)malloc(sizeof(Vertex)*vertices);
    

    bool *l=(bool *)malloc(sizeof(bool)*vertices);
    memset(l,0,sizeof(bool)*5);

    Vertex *d_gpu;
    hipMalloc(&d_gpu,sizeof(Vertex)*vertices);
    hipMemset(d_gpu, 0, sizeof(Vertex)*vertices);

    bool *l_gpu;
    hipMalloc(&l_gpu,sizeof(bool)*vertices);
    hipMemset(l_gpu,0,sizeof(bool)*vertices);

    bool *update_gpu;
    hipMalloc(&update_gpu,sizeof(bool));
    hipMemset(update_gpu, 0, sizeof(bool));

    bool *update_host = (bool*)malloc(sizeof(bool));
    *update_host=true;

    

    while(*update_host){
        hipMemset(update_gpu, false, sizeof(bool));
        find_leaf_kernel<<<(((vertices+1023)/1024)),1024>>>(l_gpu,vertices,d_gpu,g.cct,g.idx);
        hipMemcpy(l, l_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);
        Compute<<<((vertices+1023)/1024),1024>>>(round,g.cct,g.idx,l_gpu,d_gpu,vertices,update_gpu);
        round++;
        hipMemcpy(update_host, update_gpu, sizeof(bool), hipMemcpyDeviceToHost);
    }


    hipMemcpy(d_host, d_gpu, sizeof(Vertex)*vertices, hipMemcpyDeviceToHost);

    hipFree(d_gpu);
    hipFree(l_gpu);
    hipFree(update_gpu);
   
    return d_host;

    

}
#endif
