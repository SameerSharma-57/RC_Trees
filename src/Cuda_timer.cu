
#include <hip/hip_runtime.h>


class CUDA_timer{
    hipEvent_t _start,_end;
    bool recording;

    public: 
        CUDA_timer(){
            hipEventCreate(&_start);
            hipEventCreate(&_end);
            recording=false;
        }

        void start(){
            recording=true;
            hipEventRecord(_start);
        }

        void stop(){
            recording=false;
            hipEventRecord(_end);
        }

        bool isRecording(){
            return recording;
        }

        float time_elapsed_milliseconds(){
            float milliseconds = 0;

            hipEventElapsedTime(&milliseconds, _start, _end);
            return milliseconds;
        }

        float time_elapsed(){
            float milliseconds = this->time_elapsed_milliseconds();
            milliseconds/=1000;
            return milliseconds;
        }
};