#ifndef CSR
#define CSR

#include <cstddef>
#include <iostream>
using namespace std;

typedef unsigned int Vertex;
typedef unsigned long Weight;

template <class T> void Print_array(const T *array, const int size)
{
    for (int i = 0; i < size; i++)
    {
        cout << array[i] << " ";
    }
    cout << endl;
}




template<class T>
bool compareArr(T* a,T* b,Vertex size){
    bool out=true;
    for (int i = 0; i < size; i++)
    {
        if(a[i]!=b[i]){
            out=false;
            break;
        }
    }

    return out;
    
}
struct CSR_mat
{
    Weight *nnz;
    Vertex *idx;
    Vertex *cct;

    Vertex vertex_count;
    Vertex edge_count;

    bool OnCPU;

    Vertex Get_Vertex_count()const{
        return vertex_count;

    }
    Vertex Get_edge_count()const{
        return edge_count;
    }
    void Allocate(const Vertex n_vertices, const Vertex n_edges,
                  bool cpu = true)
    {

        vertex_count = n_vertices;
        edge_count = n_edges;
        OnCPU = cpu;
        if (cpu)
        {

            nnz = (Weight *)malloc(sizeof(Weight) * 2 * n_edges);
            idx = (Vertex *)malloc(sizeof(Vertex) * 2 * n_edges);
            cct = (Vertex *)malloc(sizeof(Vertex) * (n_vertices + 1));
        }

        else
        {
            hipMalloc(&nnz, sizeof(Weight) * 2 * n_edges);
            hipMalloc(&idx, sizeof(Vertex) * 2 * n_edges);
            hipMalloc(&cct, sizeof(Vertex) * (n_vertices + 1));
        }
    }

    void Deallocate()
    {
        if (OnCPU)
        {
            if (nnz != nullptr)
            {
                free(nnz);
            }
            if (idx != nullptr)
            {
                free(idx);
            }
            if (cct != nullptr)
            {
                free(cct);
            }
        }

        else
        {
            if (nnz != nullptr)
            {
                hipFree(nnz);
            }
            if (idx != nullptr)
            {
                hipFree(idx);
            }
            if (cct != nullptr)
            {
                hipFree(cct);
            }
        }
    }



    void print_mat(){
        Print_array(nnz,2*edge_count);
        cout<<endl;
        Print_array(idx, 2*edge_count);cout<<endl;
        Print_array(cct, vertex_count+1);cout<<endl;

    }
};

#endif